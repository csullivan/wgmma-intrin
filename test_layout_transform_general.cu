#include "layout_transform.cuh"

int main() {
    int m = 16;
    int k = 16; 
    int size = m * k;

    // Define rank and shapes
    int rank = 4;
    int32_t h_input_shape[MAX_RANK] = { m / 8, 8, k / 8, 8 };
    int32_t h_axes_order[MAX_RANK] = { 0, 2, 1, 3 };
    // int32_t h_axes_order[MAX_RANK] = { 2, 0, 3, 1 };

    float* h_in = (float*)malloc(size * sizeof(float));
    float* h_out_scatter = (float*)malloc(size * sizeof(float));
    float* h_out_gather = (float*)malloc(size * sizeof(float));

    for (int i = 0; i < size; i++) {
        h_in[i] = i;
    }

    float *d_in, *d_out;
    hipMalloc(&d_in, size * sizeof(float));
    hipMalloc(&d_out, size * sizeof(float));

    hipMemcpy(d_in, h_in, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch scatter transform
    launch_transform(d_in, d_out, h_input_shape, h_axes_order, rank, true);

    hipMemcpy(h_out_scatter, d_out, size * sizeof(float), hipMemcpyDeviceToHost);

    // Launch gather transform
    launch_transform(d_in, d_out, h_input_shape, h_axes_order, rank, false);

    hipMemcpy(h_out_gather, d_out, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    printf("Scatter Transform Output:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            printf("%.0f ", h_out_scatter[i * k + j]);
        }
        printf("\n");
    }

    printf("\nGather Transform Output:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            printf("%.0f ", h_out_gather[i * k + j]);
        }
        printf("\n");
    }

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out_scatter);
    free(h_out_gather);

    return 0;
}
