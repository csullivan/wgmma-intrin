#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_RANK 8

// Scatter version: Threads map over the input array
__global__ void layout_transform_scatter(float* input, float* output, int total_elements, int rank, int32_t* input_strides, int32_t* output_strides, int32_t* axes_order) {
    int input_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (input_idx >= total_elements) return;

    int input_coords[MAX_RANK];
    int output_coords[MAX_RANK];

    // Compute input_coords from input_idx
    int idx = input_idx;
    for (int i = 0; i < rank; i++) {
        input_coords[i] = idx / input_strides[i];
        idx = idx % input_strides[i];
    }

    // Map input_coords to output_coords via axes_order
    for (int i = 0; i < rank; i++) {
        output_coords[i] = input_coords[axes_order[i]];
    }

    // Compute output_idx
    int output_idx = 0;
    for (int i = 0; i < rank; i++) {
        output_idx += output_coords[i] * output_strides[i];
    }

    // Copy data
    output[output_idx] = input[input_idx];
}

// Gather version: Threads map over the output array
__global__ void layout_transform_gather(float* input, float* output, int total_elements, int rank, int32_t* input_strides, int32_t* output_strides, int32_t* axes_order_inv) {
    int output_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (output_idx >= total_elements) return;

    int output_coords[MAX_RANK];
    int input_coords[MAX_RANK];

    // Compute output_coords from output_idx
    int idx = output_idx;
    for (int i = 0; i < rank; i++) {
        output_coords[i] = idx / output_strides[i];
        idx = idx % output_strides[i];
    }

    // Map output_coords to input_coords via axes_order_inv
    for (int i = 0; i < rank; i++) {
        input_coords[i] = output_coords[axes_order_inv[i]];
    }

    // Compute input_idx
    int input_idx = 0;
    for (int i = 0; i < rank; i++) {
        input_idx += input_coords[i] * input_strides[i];
    }

    // Copy data
    output[output_idx] = input[input_idx];
}

void launch_transform(float* d_in, float* d_out, int32_t* h_input_shape, int32_t* h_axes_order, int rank, bool scatter) {
    int total_elements = 1;
    for (int i = 0; i < rank; i++) {
        total_elements *= h_input_shape[i];
    }

    int32_t h_input_strides[MAX_RANK];
    int32_t h_output_shape[MAX_RANK];
    int32_t h_output_strides[MAX_RANK];
    int32_t h_axes_order_inv[MAX_RANK];

    // Compute input_strides
    h_input_strides[rank - 1] = 1;
    for (int i = rank - 2; i >= 0; i--) {
        h_input_strides[i] = h_input_strides[i + 1] * h_input_shape[i + 1];
    }

    // Compute output_shape
    for (int i = 0; i < rank; i++) {
        h_output_shape[i] = h_input_shape[h_axes_order[i]];
    }

    // Compute output_strides
    h_output_strides[rank - 1] = 1;
    for (int i = rank - 2; i >= 0; i--) {
        h_output_strides[i] = h_output_strides[i + 1] * h_output_shape[i + 1];
    }

    // Compute axes_order_inv
    for (int i = 0; i < rank; i++) {
        h_axes_order_inv[h_axes_order[i]] = i;
    }

    // Allocate device memory for strides and axes_order
    int32_t* d_input_strides;
    int32_t* d_output_strides;
    int32_t* d_axes_order;
    int32_t* d_axes_order_inv;

    hipMalloc(&d_input_strides, rank * sizeof(int32_t));
    hipMalloc(&d_output_strides, rank * sizeof(int32_t));
    hipMalloc(&d_axes_order, rank * sizeof(int32_t));
    hipMalloc(&d_axes_order_inv, rank * sizeof(int32_t));

    hipMemcpy(d_input_strides, h_input_strides, rank * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_output_strides, h_output_strides, rank * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_axes_order, h_axes_order, rank * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_axes_order_inv, h_axes_order_inv, rank * sizeof(int32_t), hipMemcpyHostToDevice);

    // Launch kernel
    int threads_per_block = 256;
    int num_blocks = (total_elements + threads_per_block - 1) / threads_per_block;

    if (scatter) {
        layout_transform_scatter<<<num_blocks, threads_per_block>>>(d_in, d_out, total_elements, rank, d_input_strides, d_output_strides, d_axes_order);
    } else {
        layout_transform_gather<<<num_blocks, threads_per_block>>>(d_in, d_out, total_elements, rank, d_input_strides, d_output_strides, d_axes_order_inv);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    // Clean up
    hipFree(d_input_strides);
    hipFree(d_output_strides);
    hipFree(d_axes_order);
    hipFree(d_axes_order_inv);
}

int main() {
    int m = 16;
    int k = 16; 
    int size = m * k;

    // Define rank and shapes
    int rank = 4;
    int32_t h_input_shape[MAX_RANK] = { m / 8, 8, k / 8, 8 };
    int32_t h_axes_order[MAX_RANK] = { 0, 2, 1, 3 };

    float* h_in = (float*)malloc(size * sizeof(float));
    float* h_out_scatter = (float*)malloc(size * sizeof(float));
    float* h_out_gather = (float*)malloc(size * sizeof(float));

    for (int i = 0; i < size; i++) {
        h_in[i] = i;
    }

    float *d_in, *d_out;
    hipMalloc(&d_in, size * sizeof(float));
    hipMalloc(&d_out, size * sizeof(float));

    hipMemcpy(d_in, h_in, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch scatter transform
    launch_transform(d_in, d_out, h_input_shape, h_axes_order, rank, true);

    hipMemcpy(h_out_scatter, d_out, size * sizeof(float), hipMemcpyDeviceToHost);

    // Launch gather transform
    launch_transform(d_in, d_out, h_input_shape, h_axes_order, rank, false);

    hipMemcpy(h_out_gather, d_out, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    printf("Scatter Transform Output:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            printf("%.0f ", h_out_scatter[i * k + j]);
        }
        printf("\n");
    }

    printf("\nGather Transform Output:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            printf("%.0f ", h_out_gather[i * k + j]);
        }
        printf("\n");
    }

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out_scatter);
    free(h_out_gather);

    return 0;
}
